#include "hip/hip_runtime.h"
/*
 * Copyright (C) 2023, Inria
 * GRAPHDECO research group, https://team.inria.fr/graphdeco
 * All rights reserved.
 *
 * This software is free for non-commercial, research and evaluation use 
 * under the terms of the LICENSE.md file.
 *
 * For inquiries contact  george.drettakis@inria.fr
 */

#include "spatial.h"
#include "simple_knn.h"

//算对应点云最近点的函数
torch::Tensor
distCUDA2(const torch::Tensor& points)
{
  const int P = points.size(0);

  auto float_opts = points.options().dtype(torch::kFloat32);
  torch::Tensor means = torch::full({P}, 0.0, float_opts);

  // 采用KNN算法计算这个高斯点最近的三个高斯点，并计算出和这三个高斯点的平均距离， 这个可以用来去构建一个初始的高斯球
  SimpleKNN::knn(P, (float3*)points.contiguous().data<float>(), means.contiguous().data<float>());

  return means;
}